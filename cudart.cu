#include "hip/hip_runtime.h"
/*************************************************************************\
*   This program is free software: you can redistribute it and/or modify  *
*   it under the terms of the GNU General Public License as published by  *
*   the Free Software Foundation, either version 3 of the License, or     *
*   (at your option) any later version.                                   *
*                                                                         *
*   This program is distributed in the hope that it will be useful,       *
*   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
*   GNU General Public License for more details.                          *
*                                                                         *
*   You should have received a copy of the GNU General Public License     *
*   along with this program.  If not, see <http://www.gnu.org/licenses/>. *
*                                                                         *
*   cudart (c) 2008 Erik Entrich                                          *
\*************************************************************************/


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

struct vector3d
{
	float x, y, z;
};

struct rgbcolor
{
	float r, g, b;
};

struct object
{
	int type;
	struct vector3d pos;
	float rad;
	struct vector3d n;
	struct rgbcolor c;
	float e;
	float d, g;
};

#define PI 3.14159265358979323846f

#define TYPE_SPHERE 1
#define TYPE_PLANE 2

#define THREADCOUNT 16
#define TILE_width 20
#define TILE_height 20

struct object *objects;
struct object *device_objects;
int objectcount;
struct rgbcolor *imagedata;
struct rgbcolor *device_imagedata;
unsigned int *randseeds;
unsigned int *device_randseeds;
struct vector3d *raydir;
struct vector3d *device_raydir;

char envmap_filename[256] = "";
struct rgbcolor *envmap;
struct rgbcolor *device_envmap;
unsigned int envmap_width, envmap_height;
__device__ unsigned int device_envmap_width, device_envmap_height;
float envmap_offset;
__device__ float device_envmap_offset;

int width, height;
int numsamples;
int maxdepth;
char filename[256];
struct rgbcolor background;

__device__ int device_samples;
__device__ int device_width, device_height;
__device__ int device_maxdepth;
__device__ int device_objectcount;
__device__ struct rgbcolor device_background;

__device__ float vec_length( struct vector3d vec )
{
	return sqrt( vec.x*vec.x + vec.y*vec.y + vec.z*vec.z );
}

__device__ struct vector3d vec_normalize( struct vector3d vec )
{
	float length;
	struct vector3d vecout;

	length = vec_length( vec );

	vecout.x = vec.x / length;
	vecout.y = vec.y / length;
	vecout.z = vec.z / length;

	return vecout;
}

__device__ float vec_dot( struct vector3d veca, struct vector3d vecb )
{
	return (veca.x*vecb.x + veca.y*vecb.y + veca.z*vecb.z);
}

__device__ struct vector3d vec_cross( struct vector3d veca, struct vector3d vecb )
{
	struct vector3d vecout;

	vecout.x = veca.y*vecb.z - veca.z*vecb.y;
	vecout.y = veca.z*vecb.x - veca.x*vecb.z;
	vecout.z = veca.x*vecb.y - veca.y*vecb.x;

	return vecout;
}

__shared__ unsigned int randseed[THREADCOUNT];
__device__ float device_random( int tid )
{
	#define MULTIPLIER  ((unsigned int) 1664525)
	#define OFFSET  ((unsigned int) 1013904223)
	#define MODULUS  ((double) 4294967296.0)
	#define MODULUS_INV ((float) (1.0 / MODULUS))
 
	unsigned int sNew = randseed[tid] * MULTIPLIER + OFFSET;
	randseed[tid] = sNew;
	float res = sNew * MODULUS_INV;
	return res;
}

int save_pfm( unsigned int width, unsigned int height, struct rgbcolor *imagedata, char filename[], int normalize )
{
	FILE *pfm;
	int x, y;
	float r, g, b, max;

	pfm=fopen( filename, "wb" );
	if( pfm != NULL )
	{
		max = 1;
		if( normalize )
		{
			for( y=0; y<height; y++ )
			for( x=0; x<width; x++ )
			{
				if( max < imagedata[x+y*width].r )
					max = imagedata[x+y*width].r;
				if( max < imagedata[x+y*width].g )
					max = imagedata[x+y*width].g;
				if( max < imagedata[x+y*width].b )
					max = imagedata[x+y*width].b;
			}
		}

		fprintf( pfm, "PF\n%i %i\n-%f\n", width, height, max );
		for( y=0; y<height; y++ )
		for( x=0; x<width; x++ )
		{
			r = imagedata[x+y*width].r / max;
			g = imagedata[x+y*width].g / max;
			b = imagedata[x+y*width].b / max;
			fwrite( &r, sizeof( float ), 1, pfm );
			fwrite( &g, sizeof( float ), 1, pfm );
			fwrite( &b, sizeof( float ), 1, pfm );
		}

		fclose( pfm );

		return 0;
	}
	else
	{
		perror( "fopen" );
		return 1;
	}
}

struct rgbcolor *load_pfm( unsigned int *width, unsigned int *height, char filename[] )
{
	FILE *pfm;
	char buffer[256];
	unsigned int x, y;
	float r, g, b, scale;

	struct rgbcolor *imagedata;

	pfm = fopen( filename, "rb+" );
	if( pfm != NULL )
	{
		fgets( buffer, sizeof( buffer ) - 1, pfm );
		if( strcmp( buffer, "PF\n" ) != 0 )
		{
			fclose( pfm );
			return NULL;
		}

		fscanf( pfm, "%u %u\n", width, height );
		fscanf( pfm, "%f\n", &scale );
		if( scale >= 0.0f )
		{
			fclose( pfm );
			return NULL;
		}

		scale *= -1.0f;

		imagedata = (struct rgbcolor*) malloc( sizeof( struct rgbcolor ) * *width * *height );

		if( imagedata == NULL )
		{
			perror( "malloc" );
			fclose( pfm );
			return NULL;
		}

		for( y=0; y<*height; y++ )
		for( x=0; x<*width; x++ )
		{
			fread( &r, sizeof( float ), 1, pfm );
			fread( &g, sizeof( float ), 1, pfm );
			fread( &b, sizeof( float ), 1, pfm );
			imagedata[x+y**width].r = r * scale;
			imagedata[x+y**width].g = g * scale;
			imagedata[x+y**width].b = b * scale;
		}

		fclose( pfm );
		return imagedata;
	}
	else
	{
		perror( "fopen" );
		return NULL;
	}
}

int build_scene( char *scenefilename )
{
	int maxobjects = 64;
	int linenum;
	char tempbuffer[64];
	FILE *scenefile;

	objectcount = 0;
	
	objects = (struct object*) malloc( sizeof( struct object ) * maxobjects );
	if( objects == NULL )
	{
		perror( "malloc" );
		return 0;
	}

	scenefile = fopen( scenefilename, "r" );
	if( scenefile == NULL )
	{
		perror( "fopen" );
		return 0;
	}

	fscanf( scenefile, "%i %i %i %i\n", &width, &height, &numsamples, &maxdepth );
	fscanf( scenefile, "%f %f %f\n", &background.r, &background.g, &background.b );
	fgets( filename, 255, scenefile );
	filename[ strlen( filename )-1 ] = '\0';

	linenum = 3;

	while( fgets( tempbuffer, 64, scenefile ) != NULL )
	{
		linenum++;

		if( ( tempbuffer[0] == '#' ) || ( tempbuffer[0] == '\n' ) )
			continue;

		if( strcmp( tempbuffer, "sphere\n" ) == 0 )
		{
			objects[objectcount].type = TYPE_SPHERE;
			if( fscanf( scenefile, "%f %f %f %f\n", &objects[objectcount].pos.x, &objects[objectcount].pos.y, &objects[objectcount].pos.z, &objects[objectcount].rad ) != 4 )
			{
				fprintf( stderr, "Error in scnefile on line %i\n", linenum );
				return 0;
			}

			if( fscanf( scenefile, "%f %f %f\n", &objects[objectcount].c.r, &objects[objectcount].c.g, &objects[objectcount].c.b ) != 3 )
			{
				fprintf( stderr, "Error in scnefile on line %i\n", linenum );
				return 0;
			}

			if( fscanf( scenefile, "%f %f %f\n", &objects[objectcount].e, &objects[objectcount].d, &objects[objectcount].g ) != 3 )
			{
				fprintf( stderr, "Error in scnefile on line %i\n", linenum );
				return 0;
			}
		}
		else
		if( strcmp( tempbuffer, "plane\n" ) == 0 )
		{
			objects[objectcount].type = TYPE_PLANE;
			if( fscanf( scenefile, "%f %f %f\n", &objects[objectcount].pos.x, &objects[objectcount].pos.y, &objects[objectcount].pos.z ) != 3 )
			{
				fprintf( stderr, "Error in scnefile on line %i\n", linenum );
				return 0;
			}

			if( fscanf( scenefile, "%f %f %f\n", &objects[objectcount].n.x, &objects[objectcount].n.y, &objects[objectcount].n.z ) != 3 )
			{
				fprintf( stderr, "Error in scnefile on line %i\n", linenum );
				return 0;
			}

			if( fscanf( scenefile, "%f %f %f\n", &objects[objectcount].c.r, &objects[objectcount].c.g, &objects[objectcount].c.b ) != 3 )
			{
				fprintf( stderr, "Error in scnefile on line %i\n", linenum );
				return 0;
			}

			if( fscanf( scenefile, "%f %f %f\n", &objects[objectcount].e, &objects[objectcount].d, &objects[objectcount].g ) != 3 )
			{
				fprintf( stderr, "Error in scnefile on line %i\n", linenum );
				return 0;
			}
		}
		else
		{
			fprintf( stderr, "Error in scnefile on line %i\n", linenum );
			return 0;
		}

		objectcount++;
	}

	fclose( scenefile );

	return 1;
}

// need to modify this function

//__global__ void render_pixel( int row, struct rgbcolor *imagedata, struct object *globalobjects, unsigned int *randseeds, struct rgbcolor *envmap )
__global__ void render_pixel( int row, struct rgbcolor *imagedata, struct object *globalobjects, struct vector3d *raydir, unsigned int *randseeds, struct rgbcolor *envmap )
{
	volatile int px, py, tx, ty, raycounter;
	//struct vector3d raydir, campos;
	struct vector3d campos;
	struct rgbcolor pcolor, contrib;

	__shared__ struct object localobjects[64];
	volatile float mindist;
	volatile int obj, depth, counter;
	volatile float t, v, x, y, z;
	struct vector3d d, n, ir, o;

	// ************************* move global to shared : ���ÿ� copy ����
	if( threadIdx.x == 0 )
	{
		for( counter=0; counter<device_objectcount; counter++ )
		{
			localobjects[counter] = globalobjects[counter];
		}
	}	

	//randseed[threadIdx.x] =  randseeds[blockIdx.x * blockDim.x + threadIdx.x];

	__syncthreads();

	px = blockIdx.x * blockDim.x + threadIdx.x;
	py = row;

	if( px >= device_width )
		return;

	pcolor.r = 0;
	pcolor.g = 0;
	pcolor.b = 0;

	// ���� pixel �� device_sample ���� ��ŭ ��
	//for( raycounter = 0; raycounter<device_samples; raycounter++ )
	//{
		// start point
		campos.x = 0;
		campos.y = 0;
		campos.z = 0;

		//int id = blockIdx.x * blockDim.x + threadIdx.x;
		//*************************ray direction calculation with pixel information. : CPU �� ������ 
		//raydir.x = ( (float)px / (float)device_width ) - 0.5f + device_random(threadIdx.x)/(float)device_width;
		//raydir.y = ( ( (float)py / (float)device_height ) - 0.5f ) * ( (float)device_height/(float)device_width) + device_random(threadIdx.x)/(float)device_height;
		//raydir.z = 1;
		raydir[px] = vec_normalize( raydir[px] );

		contrib.r = 1.0f;
		contrib.g = 1.0f;
		contrib.b = 1.0f;

		depth = 1;

		// need to check
		// one kernel/pixel --> make thread pool and get ray 
		// while �ѹ��� ray �ϳ� 
		// if color is black, stop tracing
		// *************************
		
		while ( ( depth <= device_maxdepth ) && ( ( contrib.r * 255.0f > 1.0f ) || ( contrib.g * 255.0f > 1.0f ) || ( contrib.b * 255.0f > 1.0f ) ) )
		{
			mindist = 10000.0f;
			obj = -1;
			depth++;
			
			// get nearest object
			// intersection �� �߻��� �� �� point ���
			
			for( counter=0; counter<device_objectcount; counter++ )
			{
				if( localobjects[counter].type == TYPE_SPHERE )
				{
					d.x = localobjects[counter].pos.x - campos.x;
					d.y = localobjects[counter].pos.y - campos.y;
					d.z = localobjects[counter].pos.z - campos.z;
					v = vec_dot( raydir[px], d );

					if( v - localobjects[counter].rad > mindist )
						continue;
					
					//original + direction * t = hit point 
					t = localobjects[counter].rad*localobjects[counter].rad + v*v - d.x*d.x - d.y*d.y - d.z*d.z;
					if( t < 0 )
						continue;

					t = v - sqrt( t );
					if( ( t > mindist ) || ( t < 0 ) )
						continue;

					n.x = campos.x + t*raydir[px].x - localobjects[counter].pos.x;
					n.y = campos.y + t*raydir[px].y - localobjects[counter].pos.y;
					n.z = campos.z + t*raydir[px].z - localobjects[counter].pos.z;
					n = vec_normalize( n );

					mindist = t;
					obj = counter;
				}
				
				else if( localobjects[counter].type == TYPE_PLANE )
				{
					v = vec_dot( localobjects[counter].n, raydir[px] );

					if( v >= 0 )
						continue;

					d.x = localobjects[counter].pos.x - campos.x;
					d.y = localobjects[counter].pos.y - campos.y;
					d.z = localobjects[counter].pos.z - campos.z;

					t = vec_dot( localobjects[counter].n, d ) / v;
					if( ( t > mindist ) || ( t < 0 ) )
						continue;

					n = localobjects[counter].n;

					mindist = t;
					obj = counter;
				}

			}

			if( obj != -1 )
			{
				x = campos.x + mindist*raydir[px].x;
				y = campos.y + mindist*raydir[px].y;
				z = campos.z + mindist*raydir[px].z;

				ir.x = -raydir[px].x;
				ir.y = -raydir[px].y;
				ir.z = -raydir[px].z;

				t = 2 * vec_dot( ir, n );

				raydir[px].x = t*n.x-ir.x;
				raydir[px].y = t*n.y-ir.y;
				raydir[px].z = t*n.z-ir.z;
				raydir[px] = vec_normalize( raydir[px] );

				do
				{
					o.x = (2.0f*device_random(threadIdx.x))-1.0f;
					o.y = (2.0f*device_random(threadIdx.x))-1.0f;
					o.z = (2.0f*device_random(threadIdx.x))-1.0f;
				}
				while( ( o.x*o.x + o.y*o.y + o.z*o.z > 1 ) || ( vec_dot( o, n ) <= 0 ) );

				v = (localobjects[obj].d*device_random(threadIdx.x))*localobjects[obj].g + localobjects[obj].d*(1.0f-localobjects[obj].g);

				raydir[px].x = o.x*v+raydir[px].x*(1.0f-v);
				raydir[px].y = o.y*v+raydir[px].y*(1.0f-v);
				raydir[px].z = o.z*v+raydir[px].z*(1.0f-v);
				raydir[px] = vec_normalize( raydir[px] );

				campos.x = x+0.001f*raydir[px].x;
				campos.y = y+0.001f*raydir[px].y;
				campos.z = z+0.001f*raydir[px].z;

				contrib.r *= localobjects[obj].c.r;
				contrib.g *= localobjects[obj].c.g;
				contrib.b *= localobjects[obj].c.b;

				pcolor.r += localobjects[obj].e * contrib.r;
				pcolor.g += localobjects[obj].e * contrib.g;
				pcolor.b += localobjects[obj].e * contrib.b;
			}
			else // ���� obj �� ���� �� : �������!
			{
				if( envmap == NULL ) //envmap ���� ��: ������
				{
					pcolor.r += device_background.r * contrib.r;
					pcolor.g += device_background.g * contrib.g;
					pcolor.b += device_background.b * contrib.b;
				}
				else // ���� ��
				{
					//��� �̹����� xy point
					tx = (int)( ( ( atan2( raydir[px].x, raydir[px].z ) / PI + 1.0f ) * 0.5f ) * device_envmap_width ) % device_envmap_width;
					ty = ( atan2( raydir[px].y, sqrt( raydir[px].x*raydir[px].x + raydir[px].z*raydir[px].z ) ) / PI + 0.5f ) / ( 1.0f - device_envmap_offset ) * device_envmap_height;
					
					if( ty >= device_envmap_height )
					{
						ty = device_envmap_height - 1;
					}
					
					//��� ���� contribution: ���� �������� ���� ����.
					pcolor.r += ( (float)envmap[tx+ty*device_envmap_width].r ) * contrib.r;
					pcolor.g += ( (float)envmap[tx+ty*device_envmap_width].g ) * contrib.g;
					pcolor.b += ( (float)envmap[tx+ty*device_envmap_width].b ) * contrib.b;
				}
	
				contrib.r = 0.0f;
				contrib.g = 0.0f;
				contrib.b = 0.0f;
			}
		}
	//}

	pcolor.r = pcolor.r / (float)device_samples;
	pcolor.g = pcolor.g / (float)device_samples;
	pcolor.b = pcolor.b / (float)device_samples;

/*	imagedata[px+py*device_width].r = 1.0f - exp( -pcolor.r );
	imagedata[px+py*device_width].g = 1.0f - exp( -pcolor.g );
	imagedata[px+py*device_width].b = 1.0f - exp( -pcolor.b );*/
	imagedata[px+py*device_width].r = pcolor.r;
	imagedata[px+py*device_width].g = pcolor.g;
	imagedata[px+py*device_width].b = pcolor.b;
}

void render_image( int width, int height, int samples )
{
	int row, counter;
	int tile_x, tile_y;
	int blockcount;
	int starttime;
	hipError_t error;

	blockcount = width/THREADCOUNT + ( width%THREADCOUNT == 0?0:1 );

	puts( "Allocating memory on device" );
	if( hipMalloc( (void **)&device_objects, sizeof( struct object ) * objectcount ) != hipSuccess )
	{
		printf( "Error: %s\n", hipGetErrorString( hipGetLastError() ) );
	}
	if( hipMalloc( (void **)&device_imagedata, sizeof( struct rgbcolor ) * width * height ) != hipSuccess )
	{
		printf( "Error: %s\n", hipGetErrorString( hipGetLastError() ) );
	}
	if( hipMalloc( (void **)&device_randseeds, sizeof( unsigned int ) * TILE_width *TILE_height*samples) != hipSuccess )
	{
		printf( "Error: %s\n", hipGetErrorString( hipGetLastError() ) );
	}
	if( hipMalloc( (void **)&device_raydir, sizeof( struct vector3d ) * TILE_width *TILE_height*samples) != hipSuccess )
	{
		printf( "Error: %s\n", hipGetErrorString( hipGetLastError() ) );
	}
	if( envmap != NULL )
	{
		if( hipMalloc( (void **)&device_envmap, sizeof( struct rgbcolor ) * envmap_width * envmap_height ) != hipSuccess )
		{
			printf( "Error: %s\n", hipGetErrorString( hipGetLastError() ) );
		}
	}
	else
	{
		device_envmap = NULL;
	}

	puts( "Copying data to device" );
	if( hipMemcpy( device_objects, objects, sizeof( struct object ) * objectcount, hipMemcpyHostToDevice ) != hipSuccess )
	{
		printf( "Error: %s\n", hipGetErrorString( hipGetLastError() ) );
	}
	if( hipMemcpy( device_imagedata, imagedata, sizeof( struct rgbcolor ) * width * height, hipMemcpyHostToDevice ) != hipSuccess )
	{
		printf( "Error: %s\n", hipGetErrorString( hipGetLastError() ) );
	}
	if( envmap != NULL )
	{
		if( hipMemcpy( device_envmap, envmap, sizeof( struct rgbcolor ) * envmap_width * envmap_height, hipMemcpyHostToDevice ) != hipSuccess )
		{
			printf( "Error: %s\n", hipGetErrorString( hipGetLastError() ) );
		}
	}

	hipMemcpyToSymbol(HIP_SYMBOL( device_width), &width, sizeof( width ), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( device_height), &height, sizeof( height ), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( device_samples), &numsamples, sizeof( numsamples ), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( device_maxdepth), &maxdepth, sizeof( maxdepth ), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( device_objectcount), &objectcount, sizeof( objectcount ), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( device_background), &background, sizeof( background ), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( device_envmap_width), &envmap_width, sizeof( envmap_width ), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( device_envmap_height), &envmap_height, sizeof( envmap_height ), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL( device_envmap_offset), &envmap_offset, sizeof( envmap_offset ), 0, hipMemcpyHostToDevice );

	starttime = time( NULL );

	// *************************row ó�� :  row ���� ���� tile  : tile is rather than rows
	// first priority : ray �� ���� 1 ���� �迭 ����� : tile �� �ִ� 1 ���� �迭�� cpu �� ������ ó��
	for( tile_x=0; tile_x<width/TILE_width; tile_x++ )
	{
		for( tile_y=0; tile_y<height/TILE_height; tile_y++ )
    {
      int tile_id = tile_x * (width / TILE_width) + tile_y;

			printf( "Rendering tile %i of %i\r", tile_id, (width/TILE_width)*(height/TILE_height) ); fflush( stdout );

			for( counter=0; counter<TILE_width*TILE_height*numsamples; counter++ )
			{
				randseeds[counter] = rand();
			}

			if( hipMemcpy( device_randseeds, randseeds, sizeof( unsigned int ) * TILE_width * TILE_height * numsamples, hipMemcpyHostToDevice ) != hipSuccess )
			{
				printf( "Error: %s\n", hipGetErrorString( hipGetLastError() ) );
			}

			for( int raycounter = 0; raycounter<TILE_width*TILE_height*numsamples; raycounter++ )
			{
				//*************************ray direction calculation with pixel information. : CPU �� ������ 
				raydir[raycounter].x = ( (float)raycounter / (float)width ) - 0.5f + randseeds[raycounter]/(float)width;
				raydir[raycounter].y = ( ( (float)raycounter / (float)height ) - 0.5f ) * ( (float)height/(float)width) + randseeds[raycounter]/(float)height;
				raydir[raycounter].z = 1;
				//raydir = vec_normalize( raydir );
			}

			if (hipMemcpy( device_raydir, raydir, sizeof(struct vector3d)*TILE_width*TILE_height*numsamples, hipMemcpyHostToDevice) != hipSuccess)
			{
				printf( "Error: %s\n", hipGetErrorString( hipGetLastError() ) );
			}

			// ************************* process one ray at a time -> throughput ���� : queue �� �����
			//render_pixel <<< blockcount, THREADCOUNT >>> ( tile_id, device_imagedata, device_objects, device_randseeds, device_envmap );
			render_pixel <<< blockcount, THREADCOUNT >>> ( tile_id, device_imagedata, device_objects, device_raydir, device_randseeds, device_envmap );

			error = hipGetLastError();

			if( error != hipSuccess )
			{
				printf( "Error: %s\n", hipGetErrorString( error ) );
			}
		}
	}

	puts( "\nWaiting for threads to finish" );
	hipDeviceSynchronize();

	printf( "Time taken: %is\n", time( NULL ) - starttime );

	puts( "Copying image data from device" );
	if( hipMemcpy( imagedata, device_imagedata, sizeof( struct rgbcolor ) * width * height, hipMemcpyDeviceToHost ) != hipSuccess )
	{
		printf( "Error: %s\n", hipGetErrorString( hipGetLastError() ) );
	}

	hipFree( device_objects );
	hipFree( device_imagedata );
	hipFree( device_randseeds );
	if( envmap != NULL )
	{
		hipFree( device_envmap );
	}
}

int main( int argc, char *argv[] )
{
	puts( "Siple CUDA Ray Tracer by 50m30n3" );

	if( argc != 2 )
	{
		fputs( "USAGE: cudart scenefile\n", stderr );
		return 1;
	}

	puts( "Building scene" );
	if( !build_scene( argv[1] ) )
	{
		puts( "Cannot build scene!" );
		return 1;
	}

	puts( "Allocating Data" );
	imagedata = (struct rgbcolor*) malloc( sizeof( struct rgbcolor ) * width * height );
	if( imagedata == NULL )
	{
		perror( "malloc" );
		return 1;
	}
	randseeds = (unsigned int*) malloc( sizeof( unsigned int ) * TILE_width * TILE_height * numsamples );
	if( randseeds == NULL )
	{
		perror( "malloc" );
		return 1;
	}
	raydir = (struct vector3d *)malloc(sizeof(struct vector3d) * TILE_width * TILE_height * numsamples);
	if (raydir == NULL)
	{
		perror ("malloc raydir!");
		return 1;
	}

	envmap = NULL;
	if( strlen( envmap_filename ) > 0 )
	{
		puts( "Loading Envmap" );

		envmap = load_pfm( &envmap_width, &envmap_height, envmap_filename );
		if( envmap != NULL )
		{
			printf( "Envmap %s: %ix%i pixels\n", envmap_filename, envmap_width, envmap_height );
		}
	}

	printf( "Rendering %s at %ix%i with %i samples\n", filename, width, height, numsamples );

	render_image( width, height, numsamples );

	puts( "Saving image" );
	save_pfm( width, height, imagedata, filename, 0 );

	free( objects );
	free( imagedata );

	return 0;
}

